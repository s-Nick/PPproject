#include "hip/hip_runtime.h"

#include <GraphMol/GraphMol.h>
#include <GraphMol/FileParsers/MolSupplier.h>
#include <GraphMol/FileParsers/MolWriters.h>
#include <GraphMol/FileParsers/FileParsers.h>

#include <GraphMol/RDKitBase.h>
#include <GraphMol/SmilesParse/SmilesWrite.h>
#include <RDGeneral/FileParseException.h>
#include <RDGeneral/BadFileException.h>

#include <iostream>
#include <algorithm>
#include <vector>
#include <string>
#include <stdio.h>
#include <chrono>

#include "hip/hip_math_constants.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime.h"

#include "helper.h"

#define NUM_OF_STREAMS 360

using namespace RDKit;
using namespace std;

struct max_value{
    double distance;
    int angle;
    Rotamer rt;
    atom_st* rot_mol_fst_half;
};

__global__ void compute_unit_quaternions(double4* res, double3 quaternion){

    int tid = threadIdx.x;// + blockIdx.x*gridDim.x;

    double norm;
    double x , y ,z;
    double angle;
    double sin_2 , cos_2;


    norm = norm3d(quaternion.x, quaternion.y,quaternion.z);
    if(tid < 360){
        x = quaternion.x/norm;
        y = quaternion.y/norm;
        z = quaternion.z/norm;
        angle = HIP_PI/180 * tid;
        sin_2 = sin(angle/2);
        cos_2 = cos(angle/2);
        res[tid] = make_double4(x*sin_2, y*sin_2 , z*sin_2 , cos_2);
    }

}

int main(int argc, char** argv){

    std::string mol_file = argv[1];
    //RWMol *m = Mol2FileToMol( mol_file );
    //std::shared_ptr<RDKit::ROMol>const  mol( RDKit::Mol2FileToMol( mol_file,true,false,CORINA,false ) );

    /**
     * The following initialization works with the aspirin's mol2 file provided by the Professor.
     * The declaration above works only with the file found online.
     */
    std::shared_ptr<RDKit::ROMol>const  mol( RDKit::Mol2FileToMol( mol_file,false,true,CORINA,false ) );
    /**The next Line read the molecule removing the H atoms, it reduce the number of possible rotors
     *  for the aspirin and it seems to work, but idk with others molecules, so for now I keep
     * more rotores, but with the possible right solution.
     */
    //std::shared_ptr<RDKit::ROMol> mol( RDKit::Mol2FileToMol( mol_file,true,true,CORINA,false ) );

    Graph graph = Graph(mol->getNumAtoms());
    

    auto conf = mol->getConformer();
    
    std::cout << "number of bonds: " << mol->getNumBonds() << '\n';// mol2->getNumBonds() << '\n';

    if( !mol->getRingInfo()->isInitialized() ) {
        RDKit::MolOps::findSSSR( *mol );
    }

    for( unsigned int i = 0; i < mol->getNumBonds() ; i++ ) {
        const RDKit::Bond *bond = mol->getBondWithIdx( i );
    }

    std::vector<Rotamer> rotamers;

    for( unsigned int i = 0; i < mol->getNumBonds() ; i++ ) {
        const RDKit::Bond *bond = mol->getBondWithIdx( i );
        unsigned int startingAtom, endingAtom;
        startingAtom = bond->getBeginAtomIdx();
        endingAtom = bond->getEndAtomIdx();
        graph.addEdge(startingAtom,endingAtom);
        if( mol->getRingInfo()->numBondRings( bond->getIdx() )) {
            //continue;
            std::cout <<  "Bond " << bond->getIdx() << " is in a ring " << "stAtom: " << startingAtom << " endAtom: " << endingAtom << endl;
        }
        else if(bond->getBondType() == RDKit::Bond::BondType::DOUBLE){
            //continue;
            std::cout <<  "Bond " << bond->getIdx() << " is a DOUBLE bond " << "stAtom: " << startingAtom << " endAtom: " << endingAtom << endl;
        }
        else{
            unsigned int id = bond->getIdx();
            atom_st beginAtom;
            atom_st endAtom;
            beginAtom.id = startingAtom;
            endAtom.id = endingAtom;
            auto tmp_pos = conf.getAtomPos(beginAtom.id);
            beginAtom.position = make_double3(tmp_pos[0],tmp_pos[1],tmp_pos[2]);
            tmp_pos = conf.getAtomPos(endAtom.id);
            endAtom.position = make_double3(tmp_pos[0],tmp_pos[1],tmp_pos[2]);
            Rotamer rt = Rotamer(*bond,id, beginAtom, endAtom);
            rotamers.push_back(rt);
        }
    }

    std::vector<atom_st> atoms;

    for(auto atom : mol->atoms()){
        uint id = atom->getIdx();
        auto pos_tmp = conf.getAtomPos(id);
        double3 pos = make_double3(pos_tmp[0],pos_tmp[1],pos_tmp[2]);
        atom_st at;
        at.id = id;
        at.position = pos;
        atoms.push_back(at);
    }


    max_value max_dist;
    max_dist.distance = 0;


    vector<unsigned int> first_half;
    vector<unsigned int> second_half;
    //Rotamer rt = rotamers[0];
    //vector<Rotamer> tmp_rotamers ={rotamers[0], rotamers[1]};
    auto start = std::chrono::high_resolution_clock::now();
    for(auto rt : rotamers){

        bool analize;
        graph.removeEdge(rt.getBeginAtom().id, rt.getEndingAtom().id);

        graph.DFSlinkedNode(rt.getBeginAtom().id, first_half);
        graph.DFSlinkedNode(rt.getEndingAtom().id, second_half);

        vector<atom_st> atoms_first_half;
        vector<atom_st> atoms_second_half;
        
        for(auto i: first_half)  atoms_first_half.push_back(atoms[i]);
        
        for(auto i : second_half) atoms_second_half.push_back(atoms[i]);

        max_value max_first_half;
        max_first_half.distance = 0;
        max_value max_second_half;
        max_second_half.distance = 0;

        Rotation r;
        if(atoms_first_half.size() > 1 && second_half.size() > 1){
            analize = true;
            cout << "Checking rotamer: " << rt.getBond().getIdx() << " ";
            cout << "Starting Atom: " << rt.getBeginAtom().id << " Ending Atom: " << rt.getEndingAtom().id << " ";

            cout << "number of atom in first half: " << atoms_first_half.size() << endl;

            vector<atom_st> distance_to_compute;
            double4* unit_quaternions;

            hipMallocManaged(&unit_quaternions, 2*360*sizeof(double4));

            int deviceId;
            hipGetDevice(&deviceId);

            hipMemPrefetchAsync(unit_quaternions,2*360*sizeof(double4),deviceId);

            double3 tmp_vector = rt.getVector();

            compute_unit_quaternions<<<1,360>>>(unit_quaternions,tmp_vector);

            hipDeviceSynchronize();
            
            
            double max = 0;
            double* res;
            //cout << "main line " << __LINE__ << endl;
            for(int c = 0; c < 360; c += NUM_OF_STREAMS ){
                
                vector<vector<atom_st>> rot_first_half;
                
                double3 tmp = rt.getBeginAtom().position;

                rot_first_half = r.rotate_v5(c , atoms_first_half, tmp, unit_quaternions);

                for(int rotation = 0; rotation < NUM_OF_STREAMS; rotation++){
                    //cout << "main line " << __LINE__ << endl;
                    for(int i = 0; i < atoms_first_half.size(); i++){
                        distance_to_compute.push_back(rot_first_half[rotation][i]);
                    }
                    //cout << "main line " << __LINE__ << endl;
                    for(atom_st at : atoms_second_half){
                        distance_to_compute.push_back(at);
                    }
                }

            
                res = distance_v3(distance_to_compute, atoms.size(), NUM_OF_STREAMS);
                
                for(int i = 0; i < NUM_OF_STREAMS;i++){
                    if(res[i] > max_first_half.distance) {
                        max_first_half.distance = res[i];
                        max_first_half.angle = c+i;
                        max_first_half.rt = rt;
                    }
                }
                distance_to_compute.clear();
                
                rot_first_half.clear();
            }


            printf("Computed distance for the first part,\n");
            printf("the max distance compute is %lf with angle %d around rotamer: %d\n", \
                    max_first_half.distance, max_first_half.angle,max_first_half.rt.getBond().getIdx());
            
            hipFree(unit_quaternions);
        }
        else{
            analize = false;
            printf("Checking rotamer %d ... ", rt.getBond().getIdx());
            printf("Too few atoms in the partition, rotamer not analized\n");
        }

        double total = max_first_half.distance + max_second_half.distance;

        if(total > max_dist.distance){
            max_dist.distance = total;
            max_dist.rt = max_first_half.rt;
            max_dist.angle = max_first_half.angle;
        }

        first_half.clear();
        second_half.clear();
        atoms_first_half.clear();
        atoms_second_half.clear();

        graph.addEdge(rt.getBeginAtom().id,rt.getEndingAtom().id);
        if(analize)
            printf("For Rotamer %d, the max distance computed is: %lf,\n with a first angle: %d \n",\
                 rt.getBond().getIdx(),total,max_first_half.angle);

    }
    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop-start);

    cout << "duration time[ms]: " << duration.count() << endl;

    printf("The maximum distance computed is %lf\n", max_dist.distance);
    
    printf("Computed with an angle of %d, around the rotamer %d\n",max_dist.angle,max_dist.rt.getBond().getIdx());
    
    return 0;
}